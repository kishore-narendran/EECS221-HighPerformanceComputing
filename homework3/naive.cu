#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "timer.h"
#include "cuda_utils.h"

typedef float dtype;

#define N_ (8 * 1024 * 1024)
#define MAX_THREADS 256
#define MAX_BLOCKS 64

#define MIN(x,y) ((x < y) ? x : y)


/* return the next power of 2 number that is larger than x */
unsigned int nextPow2( unsigned int x ) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

/* find out # of threads and # thread blocks for a particular kernel */
void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
  if (whichKernel < 3)
    {
      /* 1 thread per element */
      threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
      blocks = (n + threads - 1) / threads;
    }
  else
    {
      /* 1 thread per 2 elements */
      threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
      blocks = (n + (threads * 2 - 1)) / (threads * 2);
    }
  /* limit the total number of threads */
  if (whichKernel == 5)
    blocks = MIN(maxBlocks, blocks);
}

/* special type of reduction to account for floating point error */
dtype reduce_cpu(dtype *data, int n) {
  dtype sum = data[0];
  dtype c = (dtype)0.0;
  for (int i = 1; i < n; i++)
    {
      dtype y = data[i] - c;
      dtype t = sum + y;
      c = (t - sum) - y;
      sum = t;
    }
  return sum;
}

__global__ void
kernel0 (dtype *input, dtype *output, unsigned int n)
{
  __shared__  dtype scratch[MAX_THREADS];

  unsigned int bid = gridDim.x * blockIdx.y + blockIdx.x;
  unsigned int i = bid * blockDim.x + threadIdx.x;

  if(i < n) {
    scratch[threadIdx.x] = input[i]; 
  } else {
    scratch[threadIdx.x] = 0;
  }
  __syncthreads ();

  for(unsigned int s = 1; s < blockDim.x; s = s << 1) {
    if((threadIdx.x % (2 * s)) == 0) {
      scratch[threadIdx.x] += scratch[threadIdx.x + s];
    }
    __syncthreads ();
  }

  if(threadIdx.x == 0) {
    output[bid] = scratch[0];
  }
}

int 
main(int argc, char** argv)
{
  int i;

  /* data structure */
  dtype *h_idata, h_odata, h_cpu;
  dtype *d_idata, *d_odata;	

  /* timer */
  struct stopwatch_t* timer = NULL;
  long double t_kernel_0, t_cpu;

  /* which kernel are we running */
  int whichKernel;

  /* number of threads and thread blocks */
  int threads, blocks;

  int N;
  if(argc > 1) {
    N = atoi (argv[1]);
    printf("N: %d\n", N);
  } else {
    N = N_;
    printf("N: %d\n", N);
  }

  /* naive kernel */
  whichKernel = 0;
  getNumBlocksAndThreads (whichKernel, N, MAX_BLOCKS, MAX_THREADS, 
			  blocks, threads);

  /* initialize timer */
  stopwatch_init ();
  timer = stopwatch_create ();

  /* allocate memory */
  h_idata = (dtype*) malloc (N * sizeof (dtype));
  CUDA_CHECK_ERROR (hipMalloc (&d_idata, N * sizeof (dtype)));
  CUDA_CHECK_ERROR (hipMalloc (&d_odata, blocks * sizeof (dtype)));

  /* Initialize array */
  srand48(time(NULL));
  for(i = 0; i < N; i++) {
    h_idata[i] = drand48() / 100000;
  }
  CUDA_CHECK_ERROR (hipMemcpy (d_idata, h_idata, N * sizeof (dtype), 
				hipMemcpyHostToDevice));

	
  /* ================================================== */
  /* GPU kernel */
  dim3 gb(16, ((blocks + 16 - 1) / 16), 1);
  dim3 tb(threads, 1, 1);

  /* warm up */
  kernel0 <<<gb, tb>>> (d_idata, d_odata, N);
  hipDeviceSynchronize ();
	
  stopwatch_start (timer);

  /* execute kernel */
  kernel0 <<<gb, tb>>> (d_idata, d_odata, N);
  int s = blocks;
  while(s > 1) {
    threads = 0;
    blocks = 0;
    getNumBlocksAndThreads (whichKernel, s, MAX_BLOCKS, MAX_THREADS, 
			    blocks, threads);

    dim3 gb(16, (blocks + 16 - 1) / 16, 1);
    dim3 tb(threads, 1, 1);

    kernel0 <<<gb, tb>>> (d_odata, d_odata, s);

    s = (s + threads - 1) / threads;
  }
  hipDeviceSynchronize ();

  t_kernel_0 = stopwatch_stop (timer);
  fprintf (stdout, "Time to execute naive GPU reduction kernel: %Lg secs\n",
	   t_kernel_0);
  double bw = (N * sizeof(dtype)) / (t_kernel_0 * 1e9);
  fprintf (stdout, "Effective bandwidth: %.2lf GB/s\n", bw);
	
  /* copy result back from GPU */
  CUDA_CHECK_ERROR (hipMemcpy (&h_odata, d_odata, sizeof (dtype), 
				hipMemcpyDeviceToHost));
  /* ================================================== */

  /* ================================================== */
  /* CPU kernel */
  stopwatch_start (timer);
  h_cpu = reduce_cpu (h_idata, N);
  t_cpu = stopwatch_stop (timer);
  fprintf (stdout, "Time to execute naive CPU reduction: %Lg secs\n",
	   t_cpu);
  /* ================================================== */

  if(abs (h_odata - h_cpu) > 1e-5) {
    fprintf(stderr, "FAILURE: GPU: %f 	CPU: %f\n", h_odata, h_cpu);
  } else {
    printf("SUCCESS: GPU: %f 	CPU: %f\n", h_odata, h_cpu);
  }

  return 0;
}
